#include "hip/hip_runtime.h"
#include "MyApp.h"
#include "includes/GLUtils.hpp"
#include "includes/SDL_GLDebugMessageCallback.h"
#include "includes/ProgramBuilder.h"


#include ""
#include <hip/hip_runtime.h>
#include "includes/hip/hip_runtime_api.h"

//#include <__clang_cuda_builtin_vars.h>
#include <cmath>
#include <glm/common.hpp>
#include <glm/exponential.hpp>
#include <glm/ext/matrix_transform.hpp>
#include <glm/ext/quaternion_common.hpp>
#include <glm/ext/quaternion_geometric.hpp>
#include <glm/ext/quaternion_trigonometric.hpp>
#include <glm/ext/scalar_constants.hpp>
#include <glm/fwd.hpp>
#include <glm/geometric.hpp>
#include <glm/gtc/constants.hpp>
#include <glm/matrix.hpp>
#include <imgui.h>

#include <glm/trigonometric.hpp>
#include <vector>
#include <random>

CMyApp::CMyApp()
{
}

CMyApp::~CMyApp()
{
}

void CMyApp::SetupDebugCallback()
{
	// Enable and set the debug callback function if we are in debug context
	GLint context_flags;
	glGetIntegerv(GL_CONTEXT_FLAGS, &context_flags);
	if (context_flags & GL_CONTEXT_FLAG_DEBUG_BIT)
	{
		glEnable(GL_DEBUG_OUTPUT);
		glEnable(GL_DEBUG_OUTPUT_SYNCHRONOUS);
		glDebugMessageControl(GL_DONT_CARE, GL_DONT_CARE, GL_DEBUG_SEVERITY_NOTIFICATION, 0, nullptr, GL_FALSE);
		glDebugMessageControl(GL_DONT_CARE, GL_DEBUG_TYPE_DEPRECATED_BEHAVIOR, GL_DONT_CARE, 0, nullptr, GL_FALSE);
		glDebugMessageCallback(SDL_GLDebugMessageCallback, nullptr);
	}
}

void CMyApp::InitShaders()
{
	m_programNoInstanceID = glCreateProgram();
	ProgramBuilder{ m_programNoInstanceID }
		.ShaderStage(GL_VERTEX_SHADER, "Boid.vert")
		.ShaderStage(GL_FRAGMENT_SHADER, "Boid.frag")
		.Link();
}

void CMyApp::CleanShaders()
{
	glDeleteProgram(m_programNoInstanceID);
}


void CMyApp::InitGeometry()
{
	MeshObject<glm::vec2> m_BoidMeshCPU;

	// Simple triangle
	m_BoidMeshCPU.vertexArray = {
		glm::vec2( -1, 1 ),
		glm::vec2( -1,-1 ),
		glm::vec2(  1, 0 ),
	};

	m_BoidMeshCPU.indexArray =
	{
		0, 1, 2
	};

	m_BoidGPU = CreateGLObjectFromMesh( m_BoidMeshCPU, { { 0, offsetof( glm::vec2,x), 2, GL_FLOAT}});
}

void CMyApp::CleanGeometry()
{
	CleanOGLObject( m_BoidGPU );
}

void CMyApp::InitPositions()
{
	// Initializing the Boid positions and rotations 
	std::random_device r; // seed source
	std::seed_seq seeds{r(), r(), r(), r(), r(), r(), r(), r()};
	std::mt19937 mt(seeds); // random engine with seeds
	std::uniform_real_distribution<float> randOffset(-1.0f, 1.0f);
	std::uniform_real_distribution<float> randAngle(-glm::pi<float>(), glm::pi<float>());
	
	// initialize each boid with a posiotion and an angle
	for (int i = 0; i < INST_NUM; ++i)
	{
		float angle = randAngle(mt);
		m_boids[i] = Boid {
				glm::vec2(randOffset(mt), randOffset(mt)),
				glm::vec2(glm::cos(angle), glm::sin(angle)),
			};
	}

	// Allocate vectors in device memory
  checkCudaErrors( hipMalloc(&d_boids, INST_NUM * sizeof(Boid)));
  checkCudaErrors( hipMalloc(&d_sdirs, INST_NUM * sizeof(glm::vec2)));
	checkCudaErrors( hipMalloc(&d_world_matrices, INST_NUM * sizeof(glm::mat4)));
  
  checkCudaErrors( hipMemcpy(d_boids, m_boids, INST_NUM * sizeof(Boid), hipMemcpyHostToDevice));
}

bool CMyApp::Init()
{
	SetupDebugCallback();

	// Set a bluish clear color
	// glClear() will use this for clearing the color buffer.
	glClearColor(0.125f, 0.25f, 0.5f, 1.0f);

	InitShaders();
	InitGeometry();
	InitPositions();
	//InitAttributeMode();

	// Other

	glEnable(GL_CULL_FACE);	 // Enable discarding the back-facing faces.
	glCullFace(GL_BACK);     // GL_BACK: facets facing away from camera, GL_FRONT: facets facing towards the camera

	return true;
}

void CMyApp::Clean()
{
	glDeleteBuffers(1, &m_uboID);
	glDeleteBuffers(1, &m_matrixBufferID);
	
	CleanShaders();
	CleanGeometry();

	hipFree(d_boids);
	hipFree(d_sdirs);
}

void CMyApp::Update( const SUpdateInfo& updateInfo )
{
	m_ElapsedTimeInSec = updateInfo.ElapsedTimeInSec;
	m_DeltaTimeInSec = updateInfo.DeltaTimeInSec;
}

__global__ void SteerBoids(Boid* boids, glm::vec2* sdirs)
{
	int i = threadIdx.x;
	const float FOV_COS = std::cos((FOV * M_PI / 180.0f) / 2.0f);
	sdirs[i] = boids[i].dir;

	for (int j = 0; j < INST_NUM; j++)
	{

		// see if it's the same
		if (j == i) continue;

		glm::vec2 to_other = boids[j].pos - boids[i].pos;
		float dst = glm::length(to_other);

		// see if it's inside the perception radius
		if (dst > PERCEPTION_DISTANCE) continue;

		glm::vec2 to_other_normalized = to_other / dst;

		// see if it's in the field of view
		if (glm::dot(boids[i].dir, to_other_normalized) < FOV_COS) continue;


		// TODO weight functions
		sdirs[i] +=

		// Separation
		-to_other_normalized * (glm::sqrt(PERCEPTION_DISTANCE / dst - 1.0f) * 2.5f) +

		// Alignment
		boids[j].dir +

		// Cohesion
		to_other_normalized;
	}

	sdirs[i] = glm::normalize(sdirs[i]);
}


__global__ void MoveBoids(Boid* boids, glm::vec2* sdirs, glm::mat4* world_matrices, float DeltaTimeInSec)
{
	int i = threadIdx.x;
	glm::vec3 dir = glm::vec3(boids[i].dir, 0.0f);
	glm::vec3 sdir = glm::vec3(sdirs[i], 0.0f);

	// turn towards the steering direction
	float angle = glm::acos(glm::dot(dir, sdir)) * glm::min(DeltaTimeInSec * ANGULAR_VELOCITY, 1.0f);
	glm::vec3 axis = glm::cross(dir, sdir);
	if (abs(axis.z) > 0.01f) {
		glm::vec2 ndir = glm::rotate(angle, axis) * glm::vec4(dir, 1.0f);
		boids[i].dir = ndir;
	}


	// move in the new direction
	boids[i].pos += boids[i].dir * VELOCITY * DeltaTimeInSec;
	boids[i].pos.x = std::fmodf(boids[i].pos.x + 3.0f, 2.0f) - 1.0f;
	boids[i].pos.y = std::fmodf(boids[i].pos.y + 3.0f, 2.0f) - 1.0f;

	world_matrices[i] =
		glm::translate(glm::vec3(boids[i].pos, 0))
		*
		glm::rotate(atan2(boids[i].dir.y, boids[i].dir.x), glm::vec3(0, 0, 1))
		*
		glm::scale(glm::vec3(0.01));
}

void CMyApp::DrawNoInstance()
{
	glUseProgram(m_programNoInstanceID);

	glBindVertexArray(m_BoidGPU.vaoID);

	// Set steering direction for all boids in kernel
	SteerBoids<<<1, INST_NUM>>>(d_boids, d_sdirs);
  checkCudaErrors( hipGetLastError() );
  // Set new positions based on the steering directions
	MoveBoids<<<1, INST_NUM>>>(d_boids, d_sdirs, d_world_matrices, m_DeltaTimeInSec);
  checkCudaErrors( hipGetLastError() );
  checkCudaErrors( hipMemcpy(m_world_matrices, d_world_matrices, INST_NUM * sizeof(glm::mat4), hipMemcpyDeviceToHost));

	for (int i = 0; i < INST_NUM; ++i)
	{

	// TODO mat3:
		glUniformMatrix4fv( ul("world"), 1, GL_FALSE, glm::value_ptr(m_world_matrices[i]));
		glDrawElements(GL_TRIANGLES, m_BoidGPU.count, GL_UNSIGNED_INT, 0);
	}

	glBindVertexArray(0);
	glUseProgram(0);
	
	// exit(0);
}


void CMyApp::Render()
{
	// töröljük a frampuffert (GL_COLOR_BUFFER_BIT)...
	// ... és a mélységi Z puffert (GL_DEPTH_BUFFER_BIT)
	//glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glClear(GL_COLOR_BUFFER_BIT);

	DrawNoInstance();
	//DrawUboInstance();
	//DrawArrayAttrInstanced();
}

void CMyApp::RenderGUI()
{
	// ImGui::ShowDemoWindow();
	if (ImGui::Begin("Instancing"))
	{
		const float refresh_time = 0.5f;
		static float timer = 0;
		static int   frameCount = 0;
		static float fps = 0;
		static float avgFrameTime = 0.0f;

		timer += m_DeltaTimeInSec;
		++frameCount;
		if (timer > refresh_time) {
			avgFrameTime = timer / frameCount;
			fps = frameCount / timer;
			timer = 0;
			frameCount = 0;
		}
		ImGui::Text("FPS: %d", static_cast<int>(fps));
		ImGui::Text("ms %f", avgFrameTime);
	}
	ImGui::End();
}

// New window size
void CMyApp::Resize(int _w, int _h)
{
	glViewport(0, 0, _w, _h);
	m_camera.SetAspect( static_cast<float>(_w) / _h );
}

// Other SDL events
// https://wiki.libsdl.org/SDL2/SDL_Event

void CMyApp::OtherEvent( const SDL_Event& ev )
{
}
